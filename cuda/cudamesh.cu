// example1.cpp 
  
#include <stdio.h>  
#include <hip/hip_runtime.h>  
  

__global__ void cudaKernelSquareArray( float *a, int N ) {  
	//! Kernel that executes on the CUDA device to square an array.
	//! Used for simple testing.
	int idx = blockIdx.x * blockDim.x + threadIdx.x;  
	if (idx<N) a[idx] = a[idx] * a[idx];  
}  

void cudaSimpleTest() {
  float *a_h, *a_d;  // Pointer to host & device arrays  
  const int N = 20;  // Number of elements in arrays  
  size_t size = N * sizeof(float);  
  a_h = (float *)malloc(size);        // Allocate array on host  
  hipMalloc((void **) &a_d, size);   // Allocate array on device  

  // Initialize host array and copy it to CUDA device  
  for (int i=0; i<N; i++) a_h[i] = (float)i;  
  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);  

  // Do calculation on device:  
  int block_size = 4;  
  int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);  
  cudaKernelSquareArray <<< n_blocks, block_size >>> (a_d, N);  

  // Retrieve result from device and store it in host array  
  hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);  

  // Print results  
  for (int i=0; i<N; i++) printf("%d %f\n", i, a_h[i]);  

  // Cleanup  
  free(a_h); hipFree(a_d);  
}

__global__ void cudaKernelEstEdgeProps( float* coordArr, float* distArr, float* cogArr, int N ) {  
	//! Used for edges: (kernel executed on the CUDA device)
	//! 1. estimates the length of an edge (distance between two vertices A and B)
	//! 2. estimtes the center of gravity (vertex half-way between A and B)

	int idx = blockIdx.x * blockDim.x + threadIdx.x;  
	float dX, dY, dZ;

	if( idx < N ) {
		dX = coordArr[idx*6+3] - coordArr[idx*6];
		dY = coordArr[idx*6+4] - coordArr[idx*6+1];
		dZ = coordArr[idx*6+5] - coordArr[idx*6+2];
		distArr[idx] = sqrt( dX*dX + dY*dY + dZ*dZ );  
		cogArr[idx*3]   = ( coordArr[idx*6+3] + coordArr[idx*6] ) / 2;
		cogArr[idx*3+1] = ( coordArr[idx*6+4] + coordArr[idx*6+1] ) / 2;
		cogArr[idx*3+2] = ( coordArr[idx*6+5] + coordArr[idx*6+2] ) / 2;
	}
}  
  
void cudaEstimateEdgeProperties( float* coordArr, float* distArr, float* cogArr, int elementsNr ) {
	//! Used for edges: (public)
	//! 1. estimates the length of an edge (distance between two vertices A and B)
	//! 2. estimtes the center of gravity (vertex half-way between A and B)

	float* deviceCoordArr;
	float* deviceDistArr;
	float* deviceCogArr;
	const int N = elementsNr;
	size_t size = N * sizeof( float );  

	// Allocate array on device  
	hipMalloc( (void **) &deviceCoordArr, size*6 );  
	hipMalloc( (void **) &deviceDistArr, size );  
	hipMalloc( (void **) &deviceCogArr, size );  

	// Copy coordinates to CUDA device
	hipMemcpy( deviceCoordArr, coordArr, size*6, hipMemcpyHostToDevice );  
 
	// Do calculation on device:  
	int block_size = 32;  
	int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);  
	//printf( "block_size: %d\n", block_size );  
	//printf( "n_blocks:   %d\n", n_blocks );  
	cudaKernelEstEdgeProps <<< n_blocks, block_size >>> ( deviceCoordArr, deviceDistArr, deviceCogArr, N );  

	// Retrieve result from device and store it in host array  
	hipMemcpy( distArr, deviceDistArr, sizeof( float ) * N, hipMemcpyDeviceToHost );  
	hipFree( deviceCoordArr );
	hipFree( deviceDistArr );
}
